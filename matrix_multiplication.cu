
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define TILE_DIM 16

__global__ void multiplyMatrices(int* A, int* B, int* C, int A_rows, int A_cols, int B_cols) {
    int rowIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int colIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if (rowIndex < A_rows && colIndex < B_cols) {
        int aggregate = 0;
        for (int n = 0; n < A_cols; ++n) {
            aggregate += A[rowIndex * A_cols + n] * B[n * B_cols + colIndex];
        }
        C[rowIndex * B_cols + colIndex] = aggregate;
    }
}

int main() {
    int rowsOfA, colsOfA, colsOfB;
    printf("Input dimensions of matrix A (rows): ");
    scanf("%d", &rowsOfA);
    printf("Input dimensions for matrix B (columnsA x columnsB): ");
    scanf("%d %d", &colsOfA, &colsOfB);

    int *A, *B, *C;
    int *d_A, *d_B, *d_C;

    size_t sizeOfA = rowsOfA * colsOfA * sizeof(int);
    size_t sizeOfB = colsOfA * colsOfB * sizeof(int);
    size_t sizeOfC = rowsOfA * colsOfB * sizeof(int);

    A = (int*)malloc(sizeOfA);
    B = (int*)malloc(sizeOfB);
    C = (int*)malloc(sizeOfC);

    srand((unsigned int)time(NULL));
    for (int i = 0; i < rowsOfA * colsOfA; ++i) {
        A[i] = rand() % 10;
    }
    for (int i = 0; i < colsOfA * colsOfB; ++i) {
        B[i] = rand() % 10;
    }
    hipMalloc((void**)&d_A, sizeOfA);
    hipMalloc((void**)&d_B, sizeOfB);
    hipMalloc((void**)&d_C, sizeOfC);

    hipMemcpy(d_A, A, sizeOfA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeOfB, hipMemcpyHostToDevice);

    dim3 blockDims(TILE_DIM, TILE_DIM);
    dim3 gridDims((colsOfB + blockDims.x - 1) / blockDims.x, (rowsOfA + blockDims.y - 1) / blockDims.y);

    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);

    hipEventRecord(begin);

    multiplyMatrices<<<gridDims, blockDims>>>(d_A, d_B, d_C, rowsOfA, colsOfA, colsOfB);

    hipEventRecord(end);
    hipEventSynchronize(end);

    float timeElapsed = 0;
    hipEventElapsedTime(&timeElapsed, begin, end);

    hipMemcpy(C, d_C, sizeOfC, hipMemcpyDeviceToHost);

    printf("Elapsed time: %f ms\n", timeElapsed);

    hipEventDestroy(begin);
    hipEventDestroy(end);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(A);
    free(B);
    free(C);

    return 0;
}
